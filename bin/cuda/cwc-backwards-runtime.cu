#include "hip/hip_runtime.h"
#undef USE_DISPATCH // nvcc doesn't support libdispatch
extern "C" {
#include "ccv.h"
}
#include <ctype.h>
#define CASE_TESTS // so that we don't include public available methods
#include "../lib/cuda/cwc_convnet.cu"
#include "../lib/ccv_convnet.c"

extern "C" void cwc_forwards_runtime(ccv_convnet_t* convnet, ccv_array_t* categorizeds, ccv_convnet_train_param_t params)
{
	int dual_batch = params.mini_batch;
	int category_count = 1000;
	int mini_batch = dual_batch / 2;
	_cwc_convnet_alloc_reserved_both(convnet, mini_batch, 2, params.layer_params);
	cwc_convnet_context_t* context = GPU(convnet)->contexts;
	int i, device_id, other_device_id;
	int conv_layers[] = {0, 3, 6, 7, 8};
	for (device_id = 0; device_id < 2; device_id++)
		for (i = 0; i < 5; i++)
		{
			ccv_convnet_layer_t* layer = GPU(convnet)->device[device_id].layers + conv_layers[i];
			EXTRA(layer)->vary.convolutional.forward.x = 4;
			EXTRA(layer)->vary.convolutional.forward.y = 8;
			EXTRA(layer)->vary.convolutional.forward.z = 32;
			if (conv_layers[i] == 3)
			{
				EXTRA(layer)->vary.convolutional.backward.gradient.x = 4;
				EXTRA(layer)->vary.convolutional.backward.gradient.y = 6;
				EXTRA(layer)->vary.convolutional.backward.gradient.z = 24;
				EXTRA(layer)->vary.convolutional.backward.coefficient.x = 6;
				EXTRA(layer)->vary.convolutional.backward.coefficient.y = 4;
				EXTRA(layer)->vary.convolutional.backward.coefficient.z = 24;
			} else if (conv_layers[i] == 0) {
				EXTRA(layer)->vary.convolutional.backward.coefficient.x = 1;
				EXTRA(layer)->vary.convolutional.backward.coefficient.y = 3;
				EXTRA(layer)->vary.convolutional.backward.coefficient.z = 1;
			} else {
				EXTRA(layer)->vary.convolutional.backward.gradient.x = 8;
				EXTRA(layer)->vary.convolutional.backward.gradient.y = 4;
				EXTRA(layer)->vary.convolutional.backward.gradient.z = 32;
				EXTRA(layer)->vary.convolutional.backward.coefficient.x = 8;
				EXTRA(layer)->vary.convolutional.backward.coefficient.y = 4;
				EXTRA(layer)->vary.convolutional.backward.coefficient.z = 32;
			}
		}
	for (device_id = 0; device_id < params.device_count; device_id++)
		for (other_device_id = 0; other_device_id < params.device_count; other_device_id++)
			if (device_id != other_device_id)
			{
				hipSetDevice(device_id);
				hipDeviceEnablePeerAccess(other_device_id, 0);
			}
	// doing model parallelism
	for (device_id = 0; device_id < 2; device_id++)
	{
		hipSetDevice(device_id);
		_cwc_convnet_batch_formation(0, categorizeds, convnet->mean_activity, 0, 0, 0, 0, convnet->input, convnet->rows, convnet->cols, convnet->channels, category_count, 0, mini_batch, mini_batch * device_id, mini_batch, context->host[device_id].input, context->host[device_id].c);
		hipMemcpyAsync(context->device[device_id].input, context->host[device_id].input, sizeof(float) * convnet->rows * convnet->cols * convnet->channels * mini_batch, hipMemcpyHostToDevice, context->device[device_id].data_stream);
		assert(hipGetLastError() == hipSuccess);
		hipMemcpyAsync(context->device[device_id].c, context->host[device_id].c, sizeof(int) * mini_batch, hipMemcpyHostToDevice, context->device[device_id].data_stream);
		assert(hipGetLastError() == hipSuccess);
	}
	for (device_id = 0; device_id < 2; device_id++)
	{
		hipSetDevice(device_id);
		hipDeviceSynchronize();
	}
	hipSetDevice(0);
	hipEvent_t start;
	hipEvent_t stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, context->device[0].data_stream);
	_cwc_convnet_encode_impl(convnet, 2, mini_batch, 0, context);
	for (device_id = 0; device_id < 2; device_id++)
	{
		hipSetDevice(device_id);
		// do the logistic loss
		_cwc_convnet_softmax_with_logistic_loss(mini_batch, category_count, GPU(convnet)->device[device_id].forwards[convnet->count - 1] + device_id * mini_batch * category_count, context->device[device_id].c, context->device[device_id].data_stream);
	}
	_cwc_convnet_backward_propagate_error(convnet, 2, mini_batch, context);
	hipSetDevice(1);
	hipEventRecord(context->device[1].data_joint, context->device[1].data_stream);
	hipSetDevice(0);
	hipStreamWaitEvent(context->device[0].data_stream, context->device[1].data_joint, 0);
	hipEventRecord(stop, context->device[0].data_stream);
	hipEventSynchronize(stop);
	assert(hipGetLastError() == hipSuccess);
	float elapsed_time = 0;
	hipEventElapsedTime(&elapsed_time, start, stop);
	printf("dual GPU uses %f ms\n", elapsed_time);
	float *dual_out[2] = {0};
	hipHostMalloc(&dual_out[0], sizeof(float) * dual_batch * category_count);
	hipHostMalloc(&dual_out[1], sizeof(float) * dual_batch * category_count);
	float *back_out[2] = {0};
	ccv_convnet_layer_t* second_layer = convnet->layers + 1;
	int second_count = second_layer->input.matrix.rows * second_layer->input.matrix.cols * second_layer->input.matrix.channels;
	hipHostMalloc(&back_out[0], sizeof(float) * mini_batch * second_count);
	hipHostMalloc(&back_out[1], sizeof(float) * mini_batch * second_count);
	for (device_id = 0; device_id < 2; device_id++)
	{
		hipSetDevice(device_id);
		hipMemcpy(dual_out[device_id], GPU(convnet)->device[device_id].forwards[convnet->count - 1], sizeof(float) * dual_batch * category_count, hipMemcpyDeviceToHost);
		hipMemcpy(back_out[device_id], GPU(convnet)->device[device_id].backwards[1], sizeof(float) * second_count, hipMemcpyDeviceToHost);
	}
	ccv_convnet_compact(convnet);
	assert(hipGetLastError() == hipSuccess);
	// do it on one device
	device_id = 0;
	hipSetDevice(device_id);
	_cwc_convnet_alloc_reserved_both(convnet, dual_batch, 1, params.layer_params);
	assert(hipGetLastError() == hipSuccess);
	context = GPU(convnet)->contexts;
	for (i = 0; i < 5; i++)
	{
		ccv_convnet_layer_t* layer = GPU(convnet)->device[device_id].layers + conv_layers[i];
		EXTRA(layer)->vary.convolutional.forward.x = 4;
		EXTRA(layer)->vary.convolutional.forward.y = 8;
		EXTRA(layer)->vary.convolutional.forward.z = 32;
		if (conv_layers[i] == 3)
		{
			EXTRA(layer)->vary.convolutional.backward.gradient.x = 4;
			EXTRA(layer)->vary.convolutional.backward.gradient.y = 6;
			EXTRA(layer)->vary.convolutional.backward.gradient.z = 24;
			EXTRA(layer)->vary.convolutional.backward.coefficient.x = 6;
			EXTRA(layer)->vary.convolutional.backward.coefficient.y = 4;
			EXTRA(layer)->vary.convolutional.backward.coefficient.z = 24;
		} else if (conv_layers[i] == 0) {
			EXTRA(layer)->vary.convolutional.backward.coefficient.x = 1;
			EXTRA(layer)->vary.convolutional.backward.coefficient.y = 3;
			EXTRA(layer)->vary.convolutional.backward.coefficient.z = 1;
		} else {
			EXTRA(layer)->vary.convolutional.backward.gradient.x = 8;
			EXTRA(layer)->vary.convolutional.backward.gradient.y = 4;
			EXTRA(layer)->vary.convolutional.backward.gradient.z = 32;
			EXTRA(layer)->vary.convolutional.backward.coefficient.x = 8;
			EXTRA(layer)->vary.convolutional.backward.coefficient.y = 4;
			EXTRA(layer)->vary.convolutional.backward.coefficient.z = 32;
		}
	}
	_cwc_convnet_batch_formation(0, categorizeds, convnet->mean_activity, 0, 0, 0, 0, convnet->input, convnet->rows, convnet->cols, convnet->channels, category_count, 0, dual_batch, 0, dual_batch, context->host[device_id].input, context->host[device_id].c);
	hipMemcpyAsync(context->device[device_id].input, context->host[device_id].input, sizeof(float) * convnet->rows * convnet->cols * convnet->channels * dual_batch, hipMemcpyHostToDevice, context->device[device_id].data_stream);
	assert(hipGetLastError() == hipSuccess);
	hipMemcpyAsync(context->device[device_id].c, context->host[device_id].c, sizeof(int) * dual_batch, hipMemcpyHostToDevice, context->device[device_id].data_stream);
	assert(hipGetLastError() == hipSuccess);
	hipDeviceSynchronize();
	hipEventRecord(start, context->device[0].data_stream);
	_cwc_convnet_encode_impl(convnet, 1, dual_batch, 0, context);
	// do the logistic loss
	_cwc_convnet_softmax_with_logistic_loss(dual_batch, category_count, GPU(convnet)->device[device_id].forwards[convnet->count - 1], context->device[device_id].c, context->device[device_id].data_stream);
	_cwc_convnet_backward_propagate_error(convnet, 1, dual_batch, context);
	hipEventRecord(stop, context->device[0].data_stream);
	hipEventSynchronize(stop);
	elapsed_time = 0;
	hipEventElapsedTime(&elapsed_time, start, stop);
	printf("one GPU uses %f ms\n", elapsed_time);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipDeviceSynchronize();
	float* out = 0;
	hipHostMalloc(&out, sizeof(float) * dual_batch * category_count);
	hipMemcpy(out, GPU(convnet)->device[device_id].forwards[convnet->count - 1], sizeof(float) * dual_batch * category_count, hipMemcpyDeviceToHost);
	float* back = 0;
	hipHostMalloc(&back, sizeof(float) * dual_batch * second_count);
	hipMemcpy(back, GPU(convnet)->device[device_id].backwards[1], sizeof(float) * dual_batch * second_count, hipMemcpyDeviceToHost);
	ccv_convnet_free(convnet);
	int j;
	for (i = 0; i < category_count; i++)
	{
		for (j = 0; j < mini_batch; j++)
			if (fabs(out[i * dual_batch + j] - dual_out[0][i * mini_batch + j]) > 1e-6)
				printf("softmax with logistic loss doesn't match: %d %d %g %g %g\n", i, j, out[i * dual_batch + j], dual_out[0][i * mini_batch + j], dual_out[1][i * mini_batch + j]);
		for (j = 0; j < mini_batch; j++)
			if (fabs(out[i * dual_batch + mini_batch + j] - dual_out[1][category_count * mini_batch + i * mini_batch + j]) > 1e-6)
				printf("softmax with logistic loss doesn't match: %d %d %g %g %g\n", i, j + mini_batch, out[i * dual_batch + mini_batch + j], dual_out[0][category_count * mini_batch + i * mini_batch + j], dual_out[1][1000 * mini_batch + i * mini_batch + j]);
	}/*
	for (i = 0; i < second_count; i++)
	{
		for (j = 0; j < mini_batch; j++)
			if (fabs(back[i * dual_batch + j] - back_out[0][i * mini_batch + j]) > 1e-6)
				printf("the last layer of backwards propagated error doesn't match: %d %d %g %g\n", i, j, back[i * dual_batch + j], back_out[0][i * mini_batch + j]);
	}*/
	hipHostFree(dual_out[0]);
	hipHostFree(dual_out[1]);
	hipHostFree(back_out[0]);
	hipHostFree(back_out[1]);
	hipHostFree(out);
	hipHostFree(back);
}
