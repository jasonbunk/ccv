#include "hip/hip_runtime.h"
#undef USE_DISPATCH // nvcc doesn't support libdispatch
extern "C" {
#include "ccv.h"
}
#include <ctype.h>
#define CASE_TESTS // so that we don't include public available methods
#include "../lib/cuda/cwc_convnet.cu"
#include "../lib/ccv_convnet.c"

extern "C" void cwc_forwards_runtime(ccv_convnet_t* convnet, ccv_array_t* categorizeds, ccv_convnet_train_param_t params)
{
	int dual_batch = params.mini_batch;
	int mini_batch = dual_batch / 2;
	_cwc_convnet_alloc_reserved_both(convnet, mini_batch, 2, params.layer_params);
	cwc_convnet_context_t* context = GPU(convnet)->contexts;
	int i, device_id;
	int conv_layers[] = {0, 3, 6, 7, 8};
	for (device_id = 0; device_id < 2; device_id++)
		for (i = 0; i < 5; i++)
		{
			ccv_convnet_layer_t* layer = GPU(convnet)->device[device_id].layers + conv_layers[i];
			EXTRA(layer)->vary.convolutional.forward.x = 4;
			EXTRA(layer)->vary.convolutional.forward.y = 8;
			EXTRA(layer)->vary.convolutional.forward.z = 32;
		}
	_cwc_convnet_enable_peer_access(convnet, params.device_count);
	// doing model parallelism
	for (device_id = 0; device_id < 2; device_id++)
	{
		hipSetDevice(device_id);
		_cwc_convnet_batch_formation(0, categorizeds, convnet->mean_activity, 0, 0, 0, 0, convnet->input, convnet->rows, convnet->cols, convnet->channels, 1000, 0, mini_batch, mini_batch * device_id, mini_batch, context->host[device_id].input, context->host[device_id].c);
		hipMemcpyAsync(context->device[device_id].input, context->host[device_id].input, sizeof(float) * convnet->rows * convnet->cols * convnet->channels * mini_batch, hipMemcpyHostToDevice, context->device[device_id].data_stream);
		assert(hipGetLastError() == hipSuccess);
		hipMemcpyAsync(context->device[device_id].c, context->host[device_id].c, sizeof(int) * mini_batch, hipMemcpyHostToDevice, context->device[device_id].data_stream);
		assert(hipGetLastError() == hipSuccess);
	}
	for (device_id = 0; device_id < 2; device_id++)
	{
		hipSetDevice(device_id);
		hipDeviceSynchronize();
	}
	hipSetDevice(0);
	hipEvent_t start;
	hipEvent_t stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, context->device[0].data_stream);
	_cwc_convnet_encode_impl(convnet, 2, mini_batch, 0, context);
	hipSetDevice(1);
	hipEventRecord(context->device[1].data_joint, context->device[1].data_stream);
	hipSetDevice(0);
	hipStreamWaitEvent(context->device[0].data_stream, context->device[1].data_joint, 0);
	hipEventRecord(stop, context->device[0].data_stream);
	hipEventSynchronize(stop);
	float elapsed_time = 0;
	hipEventElapsedTime(&elapsed_time, start, stop);
	printf("dual GPU uses %f ms\n", elapsed_time);
	float *dual_out[2] = {0};
	hipHostMalloc(&dual_out[0], sizeof(float) * dual_batch * 1000);
	hipHostMalloc(&dual_out[1], sizeof(float) * dual_batch * 1000);
	for (device_id = 0; device_id < 2; device_id++)
	{
		hipSetDevice(device_id);
		hipMemcpy(dual_out[device_id], GPU(convnet)->device[device_id].forwards[convnet->count - 1], sizeof(float) * dual_batch * 1000, hipMemcpyDeviceToHost);
	}
	ccv_convnet_compact(convnet);
	assert(hipGetLastError() == hipSuccess);
	// do it on one device
	device_id = 0;
	hipSetDevice(device_id);
	_cwc_convnet_alloc_reserved_both(convnet, dual_batch, 1, params.layer_params);
	assert(hipGetLastError() == hipSuccess);
	context = GPU(convnet)->contexts;
	for (i = 0; i < 5; i++)
	{
		ccv_convnet_layer_t* layer = GPU(convnet)->device[device_id].layers + conv_layers[i];
		EXTRA(layer)->vary.convolutional.forward.x = 4;
		EXTRA(layer)->vary.convolutional.forward.y = 8;
		EXTRA(layer)->vary.convolutional.forward.z = 32;
	}
	_cwc_convnet_batch_formation(0, categorizeds, convnet->mean_activity, 0, 0, 0, 0, convnet->input, convnet->rows, convnet->cols, convnet->channels, 1000, 0, dual_batch, 0, dual_batch, context->host[device_id].input, context->host[device_id].c);
	hipMemcpyAsync(context->device[device_id].input, context->host[device_id].input, sizeof(float) * convnet->rows * convnet->cols * convnet->channels * dual_batch, hipMemcpyHostToDevice, context->device[device_id].data_stream);
	assert(hipGetLastError() == hipSuccess);
	hipMemcpyAsync(context->device[device_id].c, context->host[device_id].c, sizeof(int) * dual_batch, hipMemcpyHostToDevice, context->device[device_id].data_stream);
	assert(hipGetLastError() == hipSuccess);
	hipDeviceSynchronize();
	hipEventRecord(start, context->device[0].data_stream);
	_cwc_convnet_encode_impl(convnet, 1, dual_batch, 0, context);
	hipEventRecord(stop, context->device[0].data_stream);
	hipEventSynchronize(stop);
	elapsed_time = 0;
	hipEventElapsedTime(&elapsed_time, start, stop);
	printf("one GPU uses %f ms\n", elapsed_time);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipDeviceSynchronize();
	float* out = 0;
	hipHostMalloc(&out, sizeof(float) * dual_batch * 1000);
	hipMemcpy(out, GPU(convnet)->device[device_id].forwards[convnet->count - 1], sizeof(float) * dual_batch * 1000, hipMemcpyDeviceToHost);
	ccv_convnet_free(convnet);
	int j;
	for (i = 0; i < 1000; i++)
	{
		for (j = 0; j < mini_batch; j++)
			if (fabs(out[i * dual_batch + j] - dual_out[0][i * mini_batch + j]) > 1e-3)
				printf("%d %d %f %f %f\n", i, j, out[i * dual_batch + j], dual_out[0][i * mini_batch + j], dual_out[1][i * mini_batch + j]);
		for (j = 0; j < mini_batch / 2; j++)
			if (fabs(out[i * dual_batch + mini_batch + j] - dual_out[1][1000 * mini_batch + i * mini_batch + j]) > 1e-3)
				printf("%d %d %f %f %f\n", i, j + mini_batch, out[i * dual_batch + mini_batch + j], dual_out[0][1000 * mini_batch + i * mini_batch + j], dual_out[1][1000 * mini_batch + i * mini_batch + j]);
	}
	hipHostFree(dual_out[0]);
	hipHostFree(dual_out[1]);
	hipHostFree(out);
}
