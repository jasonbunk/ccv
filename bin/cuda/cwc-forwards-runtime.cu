#include "hip/hip_runtime.h"
#undef USE_DISPATCH // nvcc doesn't support libdispatch
extern "C" {
#include "ccv.h"
}
#include <ctype.h>
#define CASE_TESTS // so that we don't include public available methods
#include "../lib/cuda/cwc_convnet.cu"
#include "../lib/ccv_convnet.c"

extern "C" void cwc_forwards_runtime(ccv_convnet_t* convnet, ccv_array_t* categorizeds, ccv_convnet_train_param_t params)
{
	int dual_batch = params.mini_batch;
	int mini_batch = dual_batch / 2;
	_cwc_convnet_alloc_reserved_both(convnet, mini_batch, 2, params.layer_params);
	cwc_convnet_context_t* context = GPU(convnet)->contexts;
	int i, device_id;
	int conv_layers[] = {0, 3, 6, 7, 8};
	for (device_id = 0; device_id < 2; device_id++)
		for (i = 0; i < 5; i++)
		{
			ccv_convnet_layer_t* layer = GPU(convnet)->device[device_id].layers + conv_layers[i];
			EXTRA(layer)->vary.convolutional.forward.x = 4;
			EXTRA(layer)->vary.convolutional.forward.y = 8;
			EXTRA(layer)->vary.convolutional.forward.z = 32;
		}
	// doing model parallelism
	for (device_id = 0; device_id < 2; device_id++)
	{
		hipSetDevice(device_id);
		_cwc_convnet_batch_formation(0, categorizeds, convnet->mean_activity, 0, 0, 0, 0, convnet->input, convnet->rows, convnet->cols, convnet->channels, 1000, 0, mini_batch, mini_batch * device_id, mini_batch, context->host[device_id].input, context->host[device_id].c);
		hipMemcpyAsync(context->device[device_id].input, context->host[device_id].input, sizeof(float) * convnet->rows * convnet->cols * convnet->channels * mini_batch, hipMemcpyHostToDevice, context->device[device_id].data_stream);
		assert(hipGetLastError() == hipSuccess);
		hipMemcpyAsync(context->device[device_id].c, context->host[device_id].c, sizeof(int) * mini_batch, hipMemcpyHostToDevice, context->device[device_id].data_stream);
		assert(hipGetLastError() == hipSuccess);
	}
	for (device_id = 0; device_id < 2; device_id++)
	{
		hipSetDevice(device_id);
		hipDeviceSynchronize();
	}
	_cwc_convnet_encode_impl(convnet, 2, mini_batch, 0, context);
	float *dual_out[2] = {0};
	hipHostMalloc(&dual_out[0], sizeof(float) * dual_batch * 1000);
	hipHostMalloc(&dual_out[1], sizeof(float) * dual_batch * 1000);
	for (device_id = 0; device_id < 2; device_id++)
	{
		hipSetDevice(device_id);
		hipDeviceSynchronize();
		hipMemcpy(dual_out[device_id], GPU(convnet)->device[device_id].forwards[convnet->count - 1], sizeof(float) * dual_batch * 1000, hipMemcpyDeviceToHost);
	}
	ccv_convnet_compact(convnet);
	assert(hipGetLastError() == hipSuccess);
	// do it on one device
	device_id = 0;
	hipSetDevice(device_id);
	mini_batch = dual_batch;
	_cwc_convnet_alloc_reserved_both(convnet, mini_batch, 1, params.layer_params);
	assert(hipGetLastError() == hipSuccess);
	context = GPU(convnet)->contexts;
	for (i = 0; i < 5; i++)
	{
		ccv_convnet_layer_t* layer = GPU(convnet)->device[device_id].layers + conv_layers[i];
		EXTRA(layer)->vary.convolutional.forward.x = 4;
		EXTRA(layer)->vary.convolutional.forward.y = 8;
		EXTRA(layer)->vary.convolutional.forward.z = 32;
	}
	_cwc_convnet_batch_formation(0, categorizeds, convnet->mean_activity, 0, 0, 0, 0, convnet->input, convnet->rows, convnet->cols, convnet->channels, 1000, 0, mini_batch, 0, mini_batch, context->host[device_id].input, context->host[device_id].c);
	hipMemcpyAsync(context->device[device_id].input, context->host[device_id].input, sizeof(float) * convnet->rows * convnet->cols * convnet->channels * mini_batch, hipMemcpyHostToDevice, context->device[device_id].data_stream);
	assert(hipGetLastError() == hipSuccess);
	hipMemcpyAsync(context->device[device_id].c, context->host[device_id].c, sizeof(int) * mini_batch, hipMemcpyHostToDevice, context->device[device_id].data_stream);
	assert(hipGetLastError() == hipSuccess);
	hipDeviceSynchronize();
	_cwc_convnet_encode_impl(convnet, 1, mini_batch, 0, context);
	hipDeviceSynchronize();
	float* out = 0;
	hipHostMalloc(&out, sizeof(float) * dual_batch * 1000);
	hipMemcpy(out, GPU(convnet)->device[device_id].forwards[convnet->count - 1], sizeof(float) * mini_batch * 1000, hipMemcpyDeviceToHost);
	ccv_convnet_free(convnet);
	int j;
	for (i = 0; i < 1000; i++)
	{
		for (j = 0; j < mini_batch / 2; j++)
			if (fabs(out[i * mini_batch + j] - dual_out[0][i * (mini_batch / 2) + j]) > 1e-3)
				printf("%d %d %f %f %f\n", i, j, out[i * mini_batch + j], dual_out[0][i * (mini_batch / 2) + j], dual_out[1][i * (mini_batch / 2) + j]);
		for (j = 0; j < mini_batch / 2; j++)
			if (fabs(out[i * mini_batch + mini_batch / 2 + j] - dual_out[0][1000 * mini_batch / 2 + i * (mini_batch / 2) + j]) > 1e-3)
				printf("%d %d %f %f %f\n", i, j + mini_batch / 2, out[i * mini_batch + mini_batch / 2 + j], dual_out[0][1000 * mini_batch / 2 + i * (mini_batch / 2) + j], dual_out[1][1000 * mini_batch / 2 + i * (mini_batch / 2) + j]);
	}
	hipHostFree(dual_out[0]);
	hipHostFree(dual_out[1]);
	hipHostFree(out);
}
